#include "hip/hip_runtime.h"
#include <stdio.h>
#include "gputimer.h"

#define NUM_THREADS 1000000
#define BLOCK_WIDTH 1000
#define ARRAY_SIZE 10

__global__ void add_naive(int *arr){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    i= i%ARRAY_SIZE;
    arr[i] = arr[i]+1;
}

__global__ void add_atomic(int *arr){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    i= i%ARRAY_SIZE;
    atomicAdd(&arr[i],1);
}

int main(int argc, char **argv){
    
    GpuTimer timer;
    int h_arr[ARRAY_SIZE];
    memset(h_arr, 0, sizeof(h_arr));

    int *d_arr;
    hipMalloc((void **) &d_arr, ARRAY_SIZE*sizeof(int));
    hipMemcpy(d_arr, h_arr, ARRAY_SIZE*sizeof(int), hipMemcpyHostToDevice);

    timer.Start();
    add_atomic<<<NUM_THREADS/BLOCK_WIDTH, BLOCK_WIDTH>>>(d_arr);
    timer.Stop();

    hipMemcpy(h_arr, d_arr, ARRAY_SIZE*sizeof(int), hipMemcpyDeviceToHost);
    for(int i=0;i<ARRAY_SIZE;i++) printf("%d, ",h_arr[i]);
    printf("\nTime taken: %g ms\n", timer.Elapsed());

    hipFree(d_arr);
    return 0;
}